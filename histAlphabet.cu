
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h> // isalnum()
#include <string.h> // stricmp()
#include <stdlib.h> // exit()
#include <sys/time.h>
#define N 26
#define BLOCK 32
#define LINE_SIZE 1024

__global__ void countAlphaOnGPU (char *idata, int *sum, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // printf("%d", ('a'-'A'));
  // printf("%c", idata[idx]);
  // printf("%d ", idata[idx]-'a');
  // printf("%d ", ('a'-'A'));
  // sum[idata[idx]-'a'] += 1;
  // printf("%d ", sum[idata[idx]-'a']);
  // printf("%d ", sum[idx]);


    // if ((idata[idx] >= 'A') && (idata[idx] <= 'Z')) {
    //   atomicAdd(&sum[idata[idx]-'A'],1);
    // } else if ((idata[idx] >= 'a') && (idata[idx] <= 'z')) {
    //   atomicAdd(&sum[idata[idx]-'a'],1);
    // }
  
  for (int i = 0; i < size; i++) {
    if ((idata[size*idx+i] >= 'A') && (idata[size*idx+i] <= 'Z')) {
      atomicAdd(&sum[idata[size*idx+i]-'A'],1);
    } else if ((idata[size*idx+i] >= 'a') && (idata[size*idx+i] <= 'z')) {
      atomicAdd(&sum[idata[size*idx+i]-'a'],1);
    }
  }

}
/*
__global__ void reduction0(int *idata, int *sum) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(sum,idata[idx]);
}

void init(int *idata) {
  for (int i = 0; i < N; i++) {
    idata[i] = 1;
  }
}
*/

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


int main(void){
  FILE *fp;
  char *fp2, *idata;
  int *odata, *sum; 
  long sz;
  char *fp3;

  double iStart_cpu, iElaps_cpu;
  double iStart_cpu2, iElaps_cpu2;
  double iStart_cpu3, iElaps_cpu3;

  double iStart_cpu4, iElaps_cpu4, iStart_cpu5, iElaps_cpu5;

  iStart_cpu = cpuSecond();

	if ( (fp=fopen("book4.txt","r"))==NULL ) {
		printf("File not open...\n");
		exit(1);
  }

  // fseek(fp, 0, SEEK_SET);
  // // printf("%c", fp->_IO_read_ptr[0]);
  // for (int j = 0; j < 1000; j++) {
  //   printf("%c", fp->_IO_read_ptr[j]);
  // }
  

  fseek(fp, 0, SEEK_END);
  sz = ftell(fp);
  printf("ファイルのサイズ : %ldバイト\n", sz);
  fseek(fp, 0, SEEK_SET);

  // for (int j = 0; j < 1000; j++) {
  //   printf("%c", fp->_IO_read_ptr[j]);
  // }
  // printf("\n");

  fp2 = (char *)malloc(sz);
  // fp2 = (char *)malloc(1024*sizeof(char));
  // fp3 = (char *)malloc(sz);
  iStart_cpu4 = cpuSecond();
  int z = 0;
  while ((fp2[z] = fgetc(fp)) != EOF) { // ラインでとる or 全て一気にとる方法を考える
      z++;
  }
  iElaps_cpu4 = cpuSecond() - iStart_cpu4;

  iStart_cpu5 = cpuSecond();
  // while ( fgets(fp2, 1024, fp) != NULL ) {
  //   strcat(fp3,fp2);
  //   // for (int i = 0; fp2[i] != '\0'; i++)
  //   //   printf("%c", fp2[i]);
  // }
  iElaps_cpu5 = cpuSecond() - iStart_cpu5;
  // for (int i = 0; fp3[i] != '\0'; i++) {
  //   printf("%c", fp3[i]);
  // }

  fseek(fp, 0, SEEK_SET);

  iStart_cpu3 = cpuSecond();
  hipMalloc((void**)&idata, sz);
  hipMalloc((void**)&odata, N*sizeof(int));
  iElaps_cpu3 = cpuSecond() - iStart_cpu3;
  // // host_idata = (char *)malloc(N*sizeof(char));
  sum = (int *)malloc(N*sizeof(int));
  // init(host_idata);
  
  // cudaMemcpy(idata, fp2, sz, cudaMemcpyHostToDevice);
  hipMemcpy(idata, fp2, sz, hipMemcpyHostToDevice);

  // free(host_idata);
  memset(sum, 0, N*sizeof(int));
  // memset(sum, 0, sizeof(int));
  // sum = 0;
  hipMemcpy(odata, sum, N*sizeof(int), hipMemcpyHostToDevice);
  // cudaMemcpy(odata, &sum, sizeof(int), cudaMemcpyHostToDevice);

  // dim3 block(BLOCK, BLOCK);
  // dim3 grid(sz/100*BLOCK,1);
  
  iStart_cpu2 = cpuSecond();
  // countAlphaOnGPU<<<grid,block>>>(idata, odata);
  // countAlphaOnGPU<<<4048,1024>>>(idata, odata);
  countAlphaOnGPU<<<sz/(LINE_SIZE*BLOCK)+1,BLOCK>>>(idata, odata, LINE_SIZE);
  iElaps_cpu2 = cpuSecond() - iStart_cpu2;
  // reduction0<<<2,256>>>(idata, odata);

  hipMemcpy(sum, odata, N*sizeof(int), hipMemcpyDeviceToHost);
  // cudaMemcpy(&sum, odata, sizeof(int), cudaMemcpyDeviceToHost);

  int i = 0;
  for (char c = 'a'; c <= 'z'; c++, i++) {
    printf("%c, %d\n", c ,sum[i]);
  }

  hipFree(idata);
  hipFree(odata);
  free(sum);
  free(fp2);
  fclose(fp);

  iElaps_cpu = cpuSecond() - iStart_cpu;
  printf("Time elapsed %f sec\n", iElaps_cpu);
  printf("Time elapsed %f sec (計算部分のみ)\n", iElaps_cpu2);
  printf("Time elapsed %f sec (cudaMalloc)\n", iElaps_cpu3);
  printf("Time elapsed %f sec (1charずつ)\n", iElaps_cpu4);
  printf("Time elapsed %f sec (1行ずつ)\n", iElaps_cpu5);

  return 0;
}